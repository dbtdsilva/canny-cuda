#include "hip/hip_runtime.h"

// Based on CUDA SDK template from NVIDIA

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>

// includes, project
#include <cutil_inline.h>

#define max(a,b) (((a)>(b))?(a):(b))
#define min(a,b) (((a)<(b))?(a):(b))

#define MAX_BRIGHTNESS 255
 
// Use int instead `unsigned char' so that we can
// store negative values.
typedef int pixel_t;

// convolution of in image to out image using kernel of kn width
void convolution(const pixel_t *in, pixel_t *out, const float *kernel,
                 const int nx, const int ny, const int kn)
{
    assert(kn % 2 == 1);
    assert(nx > kn && ny > kn);
    const int khalf = kn / 2;
 
    for (int m = khalf; m < nx - khalf; m++)
        for (int n = khalf; n < ny - khalf; n++) {
            float pixel = 0.0;
            size_t c = 0;
            for (int j = -khalf; j <= khalf; j++)
                for (int i = -khalf; i <= khalf; i++) {
                    pixel += in[(n + j) * nx + m + i] * kernel[c];
                    c++;
                }
 
            out[n * nx + m] = (pixel_t)pixel;
        }
}

// determines min and max of in image
void min_max(const pixel_t *in, const int nx, const int ny, pixel_t *pmin, pixel_t *pmax)
{
    int min = INT_MAX, max = -INT_MAX;
 
        for (int m = 0; m < nx; m++)
            for (int n = 0; n < ny ; n++) {
                int pixel = in[n*nx + m];
                if (pixel < min)
                    min = pixel;
                if (pixel > max)
                    max = pixel;
                }
    *pmin = min; *pmax = max;
}
 
// normalizes inout image using min and max values
void normalize(  pixel_t *inout, 
                 const int nx, const int ny, const int kn,
                 const int min, const int max)
{
    const int khalf = kn / 2;

    for (int m = khalf; m < nx - khalf; m++)
        for (int n = khalf; n < ny - khalf; n++) {
 
            pixel_t pixel = MAX_BRIGHTNESS * ((int)inout[n * nx + m] -(float) min) / ((float)max - (float)min);
            inout[n * nx + m] = pixel;
        }
}

 
/*
 * gaussianFilter:
 * http://www.songho.ca/dsp/cannyedge/cannyedge.html
 * determine size of kernel (odd #)
 * 0.0 <= sigma < 0.5 : 3
 * 0.5 <= sigma < 1.0 : 5
 * 1.0 <= sigma < 1.5 : 7
 * 1.5 <= sigma < 2.0 : 9
 * 2.0 <= sigma < 2.5 : 11
 * 2.5 <= sigma < 3.0 : 13 ...
 * kernelSize = 2 * int(2*sigma) + 3;
 */
void gaussian_filter(const pixel_t *in, pixel_t *out,
                     const int nx, const int ny, const float sigma)
{
    const int n = 2 * (int)(2 * sigma) + 3;
    const float mean = (float)floor(n / 2.0);
    float kernel[n * n]; // variable length array
 
    fprintf(stderr, "gaussian_filter: kernel size %d, sigma=%g\n",
            n, sigma);
    size_t c = 0;
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            kernel[c] = exp(-0.5 * (pow((i - mean) / sigma, 2.0) +
                                    pow((j - mean) / sigma, 2.0)))
                        / (2 * M_PI * sigma * sigma);
            c++;
        }
 
    convolution(in, out, kernel, nx, ny, n);
    pixel_t max, min;
    min_max(out, nx, ny, &min, &max);
    normalize(out, nx, ny, n, min, max);
}

// Canny non-maximum suppression
void non_maximum_supression(const pixel_t *after_Gx, const pixel_t * after_Gy, const pixel_t *G, pixel_t *nms, 
                            const int nx, const int ny)
{
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++) {
            const int c = i + nx * j;
            const int nn = c - nx;
            const int ss = c + nx;
            const int ww = c + 1;
            const int ee = c - 1;
            const int nw = nn + 1;
            const int ne = nn - 1;
            const int sw = ss + 1;
            const int se = ss - 1;
 
            const float dir = (float)(fmod(atan2(after_Gy[c],
                                                 after_Gx[c]) + M_PI,
                                           M_PI) / M_PI) * 8;
 
            if (((dir <= 1 || dir > 7) && G[c] > G[ee] &&
                 G[c] > G[ww]) || // 0 deg
                ((dir > 1 && dir <= 3) && G[c] > G[nw] &&
                 G[c] > G[se]) || // 45 deg
                ((dir > 3 && dir <= 5) && G[c] > G[nn] &&
                 G[c] > G[ss]) || // 90 deg
                ((dir > 5 && dir <= 7) && G[c] > G[ne] &&
                 G[c] > G[sw]))   // 135 deg
                nms[c] = G[c];
            else
                nms[c] = 0;
        }
}

// edges found in first pass for nms > tmax
void first_edges(const pixel_t *nms, pixel_t *reference, 
                 const int nx, const int ny, const int tmax)
{
 
    size_t c = 1;
    for (int j = 1; j < ny - 1; j++) {
        for (int i = 1; i < nx - 1; i++) {
            if (nms[c] >= tmax) { // trace edges
                reference[c] = MAX_BRIGHTNESS;
            }
            c++;
        }
        c+=2; // because borders are not considered
    }
}

 
// edges found in after first passes for nms > tmin && neighbor is edge
void hysteresis_edges(const pixel_t *nms, pixel_t *reference, 
                      const int nx, const int ny, const int tmin, bool *pchanged)
{
    // Tracing edges with hysteresis . Non-recursive implementation.
    for (int i = 1; i < nx - 1; i++) {
        for (int j = 1; j < ny - 1; j++) {
                    size_t t = i + j * nx;

                    int nbs[8]; // neighbours
                    nbs[0] = t - nx;     // nn
                    nbs[1] = t + nx;     // ss
                    nbs[2] = t + 1;      // ww
                    nbs[3] = t - 1;      // ee
                    nbs[4] = nbs[0] + 1; // nw
                    nbs[5] = nbs[0] - 1; // ne
                    nbs[6] = nbs[1] + 1; // sw
                    nbs[7] = nbs[1] - 1; // se
 
                    if (nms[t] >= tmin && reference[t] == 0) {
                       for(int k = 0; k < 8; k++) 
                           if (reference[nbs[k]] != 0) {
                               reference[t] = MAX_BRIGHTNESS;
                               *pchanged = true;
                           }
                    }
            }
        }
}

/*
 * Links:
 * http://en.wikipedia.org/wiki/Canny_edge_detector
 * http://www.tomgibara.com/computer-vision/CannyEdgeDetector.java
 * http://fourier.eng.hmc.edu/e161/lectures/canny/node1.html
 * http://www.songho.ca/dsp/cannyedge/cannyedge.html
 *
 * Note: T1 and T2 are lower and upper thresholds.
 */

//canny edge detector code to run on the host
void cannyHost( const int *h_idata, const int w, const int h, 
                const int tmin,            // tmin canny parameter
                const int tmax,            // tmax canny parameter
                const float sigma,         // sigma canny parameter
                int * reference)
{
    const int nx = w;
    const int ny = h;
 
    pixel_t *G        = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *after_Gx = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *after_Gy = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *nms      = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
 
    if (G == NULL || after_Gx == NULL || after_Gy == NULL ||
        nms == NULL || reference == NULL) {
        fprintf(stderr, "canny_edge_detection:"
                " Failed memory allocation(s).\n");
        exit(1);
    }
 
    // Gaussian filter
    gaussian_filter(h_idata, reference, nx, ny, sigma);
 
    const float Gx[] = {-1, 0, 1,
                        -2, 0, 2,
                        -1, 0, 1};
 
    // Gradient along x
    convolution(reference, after_Gx, Gx, nx, ny, 3);
 
    const float Gy[] = { 1, 2, 1,
                         0, 0, 0,
                        -1,-2,-1};
 
    // Gradient along y
    convolution(reference, after_Gy, Gy, nx, ny, 3);
 
    // Merging gradients
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++) {
            const int c = i + nx * j;
            G[c] = (pixel_t)(hypot((double)(after_Gx[c]), (double)( after_Gy[c]) ));
        }
 
    // Non-maximum suppression, straightforward implementation.
    non_maximum_supression(after_Gx, after_Gy, G, nms, nx, ny);

    // edges with nms >= tmax
    memset(reference, 0, sizeof(pixel_t) * nx * ny);
    first_edges(nms, reference, nx, ny, tmax);

    // edges with nms >= tmin && neighbor is edge
    bool changed;
    do {
        changed = false;
        hysteresis_edges(nms, reference, nx, ny, tmin, &changed);
    } while (changed==true);
 
    free(after_Gx);
    free(after_Gy);
    free(G);
    free(nms);
}   

/* DEVICE OPERATIONS */

__global__  void convolutionPixel(pixel_t *in, float *kernel, pixel_t *out,
                    int nx, int ny, int khalf) 
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if((x >= khalf) && (x < (nx - khalf)) &&
        (y >= khalf) && (y < (ny - khalf)))
    {
        float pixel = 0.0;
        size_t c = 0;
        for(int j = -khalf; j <= khalf; j++) 
            for(int i = -khalf; i <= khalf; i++)
                pixel += in[(y - j) * nx + x - i] * kernel[c++];
        out[y * nx + x] = (pixel_t) pixel;
    }
}

// convolution of in image to out image using kernel of kn width
void convolution_device(const pixel_t *in, pixel_t *out, const float *kernel,
                 const int nx, const int ny, const int kn)
{
    assert(kn % 2 == 1);
    assert(nx > kn && ny > kn);
    const int khalf = kn / 2;
    
    int memSize = nx * ny * sizeof(pixel_t);
    int kernelSize = kn * kn * sizeof(float);

    pixel_t *devIn;
    pixel_t *devOut;
    float *devKernel;

    hipMalloc((void**) &devIn, memSize);
    hipMalloc((void**) &devOut, memSize);
    hipMalloc((void**) &devKernel, kernelSize);

    hipMemset(devOut, 0, memSize);

    hipMemcpy(devIn, in, memSize, hipMemcpyHostToDevice);
    hipMemcpy(devKernel, kernel, kernelSize, hipMemcpyHostToDevice);

	dim3 gridSize(nx / 16 , ny / 32);				
	dim3 blockSize(16, 32);				// 512 threads (x - 16, y - 32)
    
	convolutionPixel <<<gridSize, blockSize>>> (devIn, devKernel, devOut, nx, ny, khalf);
	
    hipMemcpy(out, devOut, memSize, hipMemcpyDeviceToHost);

    hipFree(devIn);
    hipFree(devOut);
    hipFree(devKernel);
}

void gaussian_filter_device(const pixel_t *in, pixel_t *out,
                     const int nx, const int ny, const float sigma)
{
    const int n = 2 * (int)(2 * sigma) + 3;
    const float mean = (float)floor(n / 2.0);
    float kernel[n * n]; // variable length array
 
    fprintf(stderr, "gaussian_filter: kernel size %d, sigma=%g\n",
            n, sigma);
    size_t c = 0;
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            kernel[c] = exp(-0.5 * (pow((i - mean) / sigma, 2.0) +
                                    pow((j - mean) / sigma, 2.0)))
                        / (2 * M_PI * sigma * sigma);
            c++;
        }
 
    convolution_device(in, out, kernel, nx, ny, n);
    pixel_t max, min;
    min_max(out, nx, ny, &min, &max);
    normalize(out, nx, ny, n, min, max);
}

// canny edge detector code to run on the GPU
void cannyDevice( const int *h_idata, const int w, const int h, 
                  const int tmin, const int tmax, 
                  const float sigma,
                  int * h_odata)
{
    const int nx = w;
    const int ny = h;
 
    pixel_t *G        = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *after_Gx = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *after_Gy = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *nms      = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
 
    if (G == NULL || after_Gx == NULL || after_Gy == NULL ||
        nms == NULL || h_odata == NULL) {
        fprintf(stderr, "canny_edge_detection:"
                " Failed memory allocation(s).\n");
        exit(1);
    }
 
    // Gaussian filter using convolution_device
    gaussian_filter_device(h_idata, h_odata, nx, ny, sigma);
 
    const float Gx[] = {-1, 0, 1,
                        -2, 0, 2,
                        -1, 0, 1};
 
    // Gradient along x
    convolution_device(h_odata, after_Gx, Gx, nx, ny, 3);
 
    const float Gy[] = { 1, 2, 1,
                         0, 0, 0,
                        -1,-2,-1};
 
    // Gradient along y
    convolution_device(h_odata, after_Gy, Gy, nx, ny, 3);
 
    // Merging gradients
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++) {
            const int c = i + nx * j;
            G[c] = (pixel_t)(hypot((double)(after_Gx[c]), (double)( after_Gy[c]) ));
        }
 
    // Non-maximum suppression, straightforward implementation.
    non_maximum_supression(after_Gx, after_Gy, G, nms, nx, ny);

    // edges with nms >= tmax
    memset(h_odata, 0, sizeof(pixel_t) * nx * ny);
    first_edges(nms, h_odata, nx, ny, tmax);

    // edges with nms >= tmin && neighbor is edge
    bool changed;
    do {
        changed = false;
        hysteresis_edges(nms, h_odata, nx, ny, tmin, &changed);
    } while (changed==true);
 
    free(after_Gx);
    free(after_Gy);
    free(G);
    free(nms);
}

// print command line format
void usage(char *command) 
{
    printf("Usage: %s [-h] [-d device] [-i inputfile] [-o outputfile] [-r referenceFile] [-w windowsize] [-t threshold]\n",command);
}

// main
int main( int argc, char** argv) 
{

    // default command line options
    int deviceId = 0;
    char *fileIn=(char *)"lena.pgm",*fileOut=(char *)"lenaOut.pgm",*referenceOut=(char *)"reference.pgm";
    int tmin = 45, tmax = 50;
    float sigma=1.0f; 

    // parse command line arguments
    int opt;
    while( (opt = getopt(argc,argv,"d:i:o:r:n:x:s:h")) !=-1)
    {
        switch(opt)
        {

            case 'd':  // device
                if(sscanf(optarg,"%d",&deviceId)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;

            case 'i': // input image filename
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }

                fileIn = strdup(optarg);
                break;
            case 'o': // output image (from device) filename 
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                fileOut = strdup(optarg);
                break;
            case 'r': // output image (from host) filename
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                referenceOut = strdup(optarg);
                break;
            case 'n': // tmin
                if(strlen(optarg)==0 || sscanf(optarg,"%d",&tmin)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 'x': // tmax
                if(strlen(optarg)==0 || sscanf(optarg,"%d",&tmax)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 's': // sigma
                if(strlen(optarg)==0 || sscanf(optarg,"%f",&sigma)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 'h': // help
                usage(argv[0]);
                exit(0);
                break;

        }
    }

    // select cuda device
    cutilSafeCall( hipSetDevice( deviceId ) );
    
    // create events to measure host canny detector time and device canny detector time
    hipEvent_t startH, stopH, startD, stopD;
    hipEventCreate(&startH);
    hipEventCreate(&stopH);
    hipEventCreate(&startD);
    hipEventCreate(&stopD);

    // allocate host memory
    int* h_idata=NULL;
    unsigned int h,w;

    //load pgm
    if (cutLoadPGMi(fileIn, (unsigned int **)&h_idata, &w, &h) != CUTTrue) {
        printf("Failed to load image file: %s\n", fileIn);
        exit(1);
    }

    // allocate mem for the result on host side
    //int* h_odata = (int*) malloc( h*w*sizeof(unsigned int));
    //int* reference = (int*) malloc( h*w*sizeof(unsigned int));
 	
    int* h_odata = (int*) calloc(h*w, sizeof(unsigned int));
    int* reference = (int*) calloc(h*w, sizeof(unsigned int));

    // detect edges at host
    hipEventRecord( startH, 0 );
    cannyHost(h_idata, w, h, tmin, tmax, sigma, reference);   
    hipEventRecord( stopH, 0 ); 
    hipEventSynchronize( stopH );

    // detect edges at GPU
    hipEventRecord( startD, 0 );
    cannyDevice(h_idata, w, h, tmin, tmax, sigma, h_odata);   
    hipEventRecord( stopD, 0 ); 
    hipEventSynchronize( stopD );
    
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    float timeH, timeD;
    hipEventElapsedTime( &timeH, startH, stopH );
    printf( "Host processing time: %f (ms)\n", timeH);
    hipEventElapsedTime( &timeD, startD, stopD );
    printf( "Device processing time: %f (ms)\n", timeD);

    // save output images
    if (cutSavePGMi(referenceOut, (unsigned int *)reference, w, h) != CUTTrue) {
        printf("Failed to save image file: %s\n", referenceOut);
        exit(1);
    }
    if (cutSavePGMi(fileOut,(unsigned int *) h_odata, w, h) != CUTTrue) {
        printf("Failed to save image file: %s\n", fileOut);
        exit(1);
    }

    // cleanup memory
    cutFree( h_idata);
    free( h_odata);
    free( reference);

    cutilDeviceReset();
}
