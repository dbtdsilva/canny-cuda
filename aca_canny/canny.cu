#include "hip/hip_runtime.h"

// Based on CUDA SDK template from NVIDIA

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>

// includes, project
#include <cutil_inline.h>

#define max(a,b) (((a)>(b))?(a):(b))
#define min(a,b) (((a)<(b))?(a):(b))

#define MAX_BRIGHTNESS 255
 
// Use int instead `unsigned char' so that we can
// store negative values.
typedef int pixel_t;

// Device constant
__constant__ int const_nx;
__constant__ int const_ny;
__constant__ int const_khalf;

// convolution of in image to out image using kernel of kn width
void convolution(const pixel_t *in, pixel_t *out, const float *kernel,
                 const int nx, const int ny, const int kn)
{
    assert(kn % 2 == 1);
    assert(nx > kn && ny > kn);
    const int khalf = kn / 2;
 
    for (int m = khalf; m < nx - khalf; m++)
        for (int n = khalf; n < ny - khalf; n++) {
            float pixel = 0.0;
            size_t c = 0;
            for (int j = -khalf; j <= khalf; j++)
                for (int i = -khalf; i <= khalf; i++) {
                    pixel += in[(n + j) * nx + m + i] * kernel[c];
                    c++;
                }
 
            out[n * nx + m] = (pixel_t)pixel;
        }
}

// determines min and max of in image
void min_max(const pixel_t *in, const int nx, const int ny, pixel_t *pmin, pixel_t *pmax)
{
    int min = INT_MAX, max = -INT_MAX;
 
        for (int m = 0; m < nx; m++)
            for (int n = 0; n < ny ; n++) {
                int pixel = in[n*nx + m];
                if (pixel < min)
                    min = pixel;
                if (pixel > max)
                    max = pixel;
                }
    *pmin = min; *pmax = max;
}
 
// normalizes inout image using min and max values
void normalize(  pixel_t *inout, 
                 const int nx, const int ny, const int kn,
                 const int min, const int max)
{
    const int khalf = kn / 2;

    for (int m = khalf; m < nx - khalf; m++)
        for (int n = khalf; n < ny - khalf; n++) {
 
            pixel_t pixel = MAX_BRIGHTNESS * ((int)inout[n * nx + m] -(float) min) / ((float)max - (float)min);
            inout[n * nx + m] = pixel;
        }
}

 
/*
 * gaussianFilter:
 * http://www.songho.ca/dsp/cannyedge/cannyedge.html
 * determine size of kernel (odd #)
 * 0.0 <= sigma < 0.5 : 3
 * 0.5 <= sigma < 1.0 : 5
 * 1.0 <= sigma < 1.5 : 7
 * 1.5 <= sigma < 2.0 : 9
 * 2.0 <= sigma < 2.5 : 11
 * 2.5 <= sigma < 3.0 : 13 ...
 * kernelSize = 2 * int(2*sigma) + 3;
 */
void gaussian_filter(const pixel_t *in, pixel_t *out,
                     const int nx, const int ny, const float sigma)
{
    const int n = 2 * (int)(2 * sigma) + 3;
    const float mean = (float)floor(n / 2.0);
    float kernel[n * n]; // variable length array
 
    fprintf(stderr, "gaussian_filter: kernel size %d, sigma=%g\n",
            n, sigma);
    size_t c = 0;
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            kernel[c] = exp(-0.5 * (pow((i - mean) / sigma, 2.0) +
                                    pow((j - mean) / sigma, 2.0)))
                        / (2 * M_PI * sigma * sigma);
            c++;
        }
 
    convolution(in, out, kernel, nx, ny, n);
    pixel_t max, min;
    min_max(out, nx, ny, &min, &max);
    normalize(out, nx, ny, n, min, max);
}

// Canny non-maximum suppression
void non_maximum_supression(const pixel_t *after_Gx, const pixel_t * after_Gy, const pixel_t *G, pixel_t *nms, 
                            const int nx, const int ny)
{
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++) {
            const int c = i + nx * j;
            const int nn = c - nx;
            const int ss = c + nx;
            const int ww = c + 1;
            const int ee = c - 1;
            const int nw = nn + 1;
            const int ne = nn - 1;
            const int sw = ss + 1;
            const int se = ss - 1;
 
            const float dir = (float)(fmod(atan2(after_Gy[c],
                                                 after_Gx[c]) + M_PI,
                                           M_PI) / M_PI) * 8;
 
            if (((dir <= 1 || dir > 7) && G[c] > G[ee] &&
                 G[c] > G[ww]) || // 0 deg
                ((dir > 1 && dir <= 3) && G[c] > G[nw] &&
                 G[c] > G[se]) || // 45 deg
                ((dir > 3 && dir <= 5) && G[c] > G[nn] &&
                 G[c] > G[ss]) || // 90 deg
                ((dir > 5 && dir <= 7) && G[c] > G[ne] &&
                 G[c] > G[sw]))   // 135 deg
                nms[c] = G[c];
            else
                nms[c] = 0;
        }
}

// edges found in first pass for nms > tmax
void first_edges(const pixel_t *nms, pixel_t *reference, 
                 const int nx, const int ny, const int tmax)
{
 
    size_t c = 1;
    for (int j = 1; j < ny - 1; j++) {
        for (int i = 1; i < nx - 1; i++) {
            if (nms[c] >= tmax) { // trace edges
                reference[c] = MAX_BRIGHTNESS;
            }
            c++;
        }
        c+=2; // because borders are not considered
    }
}

 
// edges found in after first passes for nms > tmin && neighbor is edge
void hysteresis_edges(const pixel_t *nms, pixel_t *reference, 
                      const int nx, const int ny, const int tmin, bool *pchanged)
{
    // Tracing edges with hysteresis . Non-recursive implementation.
    for (int i = 1; i < nx - 1; i++) {
        for (int j = 1; j < ny - 1; j++) {
                    size_t t = i + j * nx;

                    int nbs[8]; // neighbours
                    nbs[0] = t - nx;     // nn
                    nbs[1] = t + nx;     // ss
                    nbs[2] = t + 1;      // ww
                    nbs[3] = t - 1;      // ee
                    nbs[4] = nbs[0] + 1; // nw
                    nbs[5] = nbs[0] - 1; // ne
                    nbs[6] = nbs[1] + 1; // sw
                    nbs[7] = nbs[1] - 1; // se
 
                    if (nms[t] >= tmin && reference[t] == 0) {
                       for(int k = 0; k < 8; k++) 
                           if (reference[nbs[k]] != 0) {
                               reference[t] = MAX_BRIGHTNESS;
                               *pchanged = true;
                           }
                    }
            }
        }
}

/*
 * Links:
 * http://en.wikipedia.org/wiki/Canny_edge_detector
 * http://www.tomgibara.com/computer-vision/CannyEdgeDetector.java
 * http://fourier.eng.hmc.edu/e161/lectures/canny/node1.html
 * http://www.songho.ca/dsp/cannyedge/cannyedge.html
 *
 * Note: T1 and T2 are lower and upper thresholds.
 */

//canny edge detector code to run on the host
void cannyHost( const int *h_idata, const int w, const int h, 
                const int tmin,            // tmin canny parameter
                const int tmax,            // tmax canny parameter
                const float sigma,         // sigma canny parameter
                int * reference)
{
    const int nx = w;
    const int ny = h;
 
    pixel_t *G        = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *after_Gx = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *after_Gy = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
    pixel_t *nms      = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));
 
    if (G == NULL || after_Gx == NULL || after_Gy == NULL ||
        nms == NULL || reference == NULL) {
        fprintf(stderr, "canny_edge_detection:"
                " Failed memory allocation(s).\n");
        exit(1);
    }
 
    // Gaussian filter
    gaussian_filter(h_idata, reference, nx, ny, sigma);
 
    const float Gx[] = {-1, 0, 1,
                        -2, 0, 2,
                        -1, 0, 1};
 
    // Gradient along x
    convolution(reference, after_Gx, Gx, nx, ny, 3);
 
    const float Gy[] = { 1, 2, 1,
                         0, 0, 0,
                        -1,-2,-1};
 
    // Gradient along y
    convolution(reference, after_Gy, Gy, nx, ny, 3);
 
    // Merging gradients
    for (int i = 1; i < nx - 1; i++)
        for (int j = 1; j < ny - 1; j++) {
            const int c = i + nx * j;
            G[c] = (pixel_t)(hypot((double)(after_Gx[c]), (double)( after_Gy[c]) ));
        }
 
    // Non-maximum suppression, straightforward implementation.
    non_maximum_supression(after_Gx, after_Gy, G, nms, nx, ny);

    // edges with nms >= tmax
    memset(reference, 0, sizeof(pixel_t) * nx * ny);
    first_edges(nms, reference, nx, ny, tmax);

    // edges with nms >= tmin && neighbor is edge
    bool changed;
    do {
        changed = false;
        hysteresis_edges(nms, reference, nx, ny, tmin, &changed);
    } while (changed==true);
 
    free(after_Gx);
    free(after_Gy);
    free(G);
    free(nms);
}   

/* DEVICE OPERATIONS */

__global__  void convolution_kernel(const pixel_t *in, const float *kernel, pixel_t *out) 
{
    int x = threadIdx.x + blockIdx.x * blockDim.x + const_khalf;
    int y = threadIdx.y + blockIdx.y * blockDim.y + const_khalf;

    if((x < (const_nx - const_khalf)) && (y < (const_ny - const_khalf)))
    {
        const int width = 18;
        const int height = 34;
        const int size = width * height;
        const bool vLimit = (y == const_ny-2);
        const bool hLimit = (x == const_nx-2);
        __shared__ pixel_t subMatrix[size];

        int sub_x = threadIdx.x + const_khalf;
        int sub_y = threadIdx.y + const_khalf;

        if(sub_x == 1 && sub_y == 1)
            subMatrix[(sub_y-1)*width + sub_x-1] = in[(y-1)*const_nx + x-1];
        else if(sub_x == 1 && (vLimit || sub_y == height-2))
            subMatrix[(sub_y+1)*width + sub_x-1] = in[(y+1)*const_nx + x-1];
        else if((hLimit || sub_x == width-2) && sub_y == 1)
            subMatrix[(sub_y-1)*width + sub_x+1] = in[(y-1)*const_nx + x+1];
        else if((hLimit || sub_x == width-2) && (vLimit || sub_y == height-2))
            subMatrix[(sub_y+1)*width + sub_x+1] = in[(y+1)*const_nx + x+1];

        if(sub_x == 1)
            subMatrix[sub_y*width + sub_x-1] = in[y*const_nx + x-1];
        else if(hLimit || sub_x == width-2)
            subMatrix[sub_y*width + sub_x+1] = in[y*const_nx + x+1];

        if(sub_y == 1)
            subMatrix[(sub_y-1)*width + sub_x] = in[(y-1)*const_nx + x];
        else if(vLimit || sub_y == height-2)
            subMatrix[(sub_y+1)*width + sub_x] = in[(y+1)*const_nx + x];

        subMatrix[sub_y*width + sub_x] = in[y*const_nx + x];

        __syncthreads();

        float pixel = 0.0;
        size_t c = 0;
        for(int j = -const_khalf; j <= const_khalf; j++) 
            for(int i = -const_khalf; i <= const_khalf; i++)
                pixel += subMatrix[(sub_y+j)*width + sub_x+i] * kernel[c++];
        out[y*const_nx + x] = (pixel_t) pixel;
    }
}

// convolution of in image to out image using kernel of kn width
void convolution_device(const pixel_t *in, pixel_t *out, const float *kernel,
                 const int nx, const int ny, const int kn)
{
    assert(kn % 2 == 1);
    assert(nx > kn && ny > kn);
    
    const int khalf = kn / 2;
    
    hipMemcpyToSymbol(HIP_SYMBOL(const_khalf), &khalf, sizeof(int));

    dim3 gridSize(ceil((nx - 2*khalf)/ 16.0), ceil((ny - 2*khalf)/ 32.0));              
    dim3 blockSize(16, 32);             // 512 threads (x - 16, y - 32)
    
    convolution_kernel <<<gridSize, blockSize>>> (in, kernel, out);
}

__global__  void non_maximum_supression_kernel(const pixel_t *afterGx, const pixel_t *afterGy,
                            const pixel_t *G, pixel_t *nms)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int y = threadIdx.y + blockIdx.y * blockDim.y + 1;
    
    if((x < (const_nx - 1)) && (y < (const_ny - 1)))
    {
        const int width = 18;
        const int height = 34;
        const int size = width * height;
        const bool vLimit = (y == const_ny-2);
        const bool hLimit = (x == const_nx-2);
        __shared__ pixel_t subMatrix[size];

        int sub_x = threadIdx.x + 1;
        int sub_y = threadIdx.y + 1;

        if(sub_x == 1 && sub_y == 1)
            subMatrix[(sub_y-1)*width + sub_x-1] = G[(y-1)*const_nx + x-1];
        else if(sub_x == 1 && (vLimit || sub_y == height-2))
            subMatrix[(sub_y+1)*width + sub_x-1] = G[(y+1)*const_nx + x-1];
        else if((hLimit || sub_x == width-2) && sub_y == 1)
            subMatrix[(sub_y-1)*width + sub_x+1] = G[(y-1)*const_nx + x+1];
        else if((hLimit || sub_x == width-2) && (vLimit || sub_y == height-2))
            subMatrix[(sub_y+1)*width + sub_x+1] = G[(y+1)*const_nx + x+1];

        if(sub_x == 1)
            subMatrix[sub_y*width + sub_x-1] = G[y*const_nx + x-1];
        else if(hLimit || sub_x == width-2)
            subMatrix[sub_y*width + sub_x+1] = G[y*const_nx + x+1];

        if(sub_y == 1)
            subMatrix[(sub_y-1)*width + sub_x] = G[(y-1)*const_nx + x];
        else if(vLimit || sub_y == height-2)
            subMatrix[(sub_y+1)*width + sub_x] = G[(y+1)*const_nx + x];

        subMatrix[sub_y*width + sub_x] = G[y*const_nx + x];

        int c = y*const_nx + x;
        int sub_c = sub_x + width*sub_y;
        int nn = c - width;
        int ss = c + width;
        int ww = c + 1;
        int ee = c - 1;
        int nw = nn + 1;
        int ne = nn - 1;
        int sw = ss + 1;
        int se = ss - 1;

        __syncthreads();

        float dir = (float) (fmod(atan2((double) afterGy[c],(double) afterGx[c]) + M_PI, M_PI) / M_PI) * 8;

        if(((dir <= 1 || dir > 7) && subMatrix[sub_c] > subMatrix[ee] && subMatrix[sub_c] > subMatrix[ww]) ||
           ((dir > 1 && dir <= 3) && subMatrix[sub_c] > subMatrix[nw] && subMatrix[sub_c] > subMatrix[se]) ||
           ((dir > 3 && dir <= 5) && subMatrix[sub_c] > subMatrix[nn] && subMatrix[sub_c] > subMatrix[ss]) ||
           ((dir > 5 && dir <= 7) && subMatrix[sub_c] > subMatrix[ne] && subMatrix[sub_c] > subMatrix[sw]))
            nms[c] = subMatrix[sub_c];
        else
            nms[c] = 0;
    }
}

// Canny non-maximum suppression
void non_maximum_supression_device(const pixel_t *after_Gx, const pixel_t * after_Gy,
                    const pixel_t *G, pixel_t *nms, const int nx, const int ny)
{
    dim3 gridSize(ceil((nx - 2)/ 16.0), ceil((ny - 2)/ 32.0));              
    dim3 blockSize(16, 32);             // 512 threads (x - 16, y - 32)
    
    non_maximum_supression_kernel <<<gridSize, blockSize>>> (after_Gx, after_Gy, G, nms);
}

__global__ void merging_gradients_kernel(const pixel_t *afterGx, const pixel_t *afterGy,
                                        pixel_t *G)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int y = threadIdx.y + blockIdx.y * blockDim.y + 1;
    
    if((x < (const_nx - 1)) && (y < (const_ny - 1)))
    {
        const int c = x + const_nx * y;
        G[c] = (pixel_t)(hypot((double)(afterGx[c]), (double)( afterGy[c]) ));
    }
}

void merging_gradients_device(const pixel_t *after_Gx, const pixel_t *after_Gy, pixel_t *G,
                            const int nx, const int ny)
{
    dim3 gridSize(ceil((nx - 2)/ 16.0), ceil((ny - 2)/ 32.0));              
    dim3 blockSize(16, 32);             // 512 threads (x - 16, y - 32)
    
    merging_gradients_kernel <<<gridSize, blockSize>>> (after_Gx, after_Gy, G);
}

// canny edge detector code to run on the GPU
void cannyDevice( const int *h_idata, const int w, const int h, 
                  const int tmin, const int tmax, 
                  const float sigma,
                  int * h_odata)
{
    const int nx = w;
    const int ny = h;
    const int memSize = nx * ny * sizeof(pixel_t);
    const int gradSize = 3 * 3 * sizeof(float);

    hipMemcpyToSymbol(HIP_SYMBOL(const_nx), &nx, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(const_ny), &ny, sizeof(int));
 
    pixel_t *nms      = (pixel_t *) calloc(nx * ny, sizeof(pixel_t));

    // cuda pointers
    pixel_t *dev_h_odata;
    pixel_t *dev_G;
    pixel_t *dev_after_Gx;
    pixel_t *dev_after_Gy;
    pixel_t *dev_nms;
    float *dev_grad;

    hipMalloc((void**) &dev_h_odata, memSize);
    hipMalloc((void**) &dev_G, memSize);
    hipMalloc((void**) &dev_after_Gx, memSize);
    hipMalloc((void**) &dev_after_Gy, memSize);
    hipMalloc((void**) &dev_nms, memSize);
    hipMalloc((void**) &dev_grad, gradSize);

    hipMemset(dev_h_odata, 0, memSize);
    hipMemset(dev_G, 0, memSize);
    hipMemset(dev_after_Gx, 0, memSize);
    hipMemset(dev_after_Gy, 0, memSize);
    hipMemset(dev_nms, 0, memSize);

    if (nms == NULL || h_odata == NULL) {
        fprintf(stderr, "canny_edge_detection:"
                " Failed memory allocation(s).\n");
        exit(1);
    }
 
    // Gaussian filter using convolution_device
    gaussian_filter(h_idata, h_odata, nx, ny, sigma);
    
    hipMemcpy(dev_h_odata, h_odata, memSize, hipMemcpyHostToDevice);

    const float Gx[] = {-1, 0, 1,
                        -2, 0, 2,
                        -1, 0, 1};
 
    hipMemcpy(dev_grad, Gx, gradSize, hipMemcpyHostToDevice);

    // Gradient along x
    convolution_device(dev_h_odata, dev_after_Gx, dev_grad, nx, ny, 3);
 
    const float Gy[] = { 1, 2, 1,
                         0, 0, 0,
                        -1,-2,-1};
 
    hipMemcpy(dev_grad, Gy, gradSize, hipMemcpyHostToDevice);

    // Gradient along y
    convolution_device(dev_h_odata, dev_after_Gy, dev_grad, nx, ny, 3);

    merging_gradients_device(dev_after_Gx, dev_after_Gy, dev_G, nx, ny);
 
    // Non-maximum suppression, straightforward implementation.
    non_maximum_supression_device(dev_after_Gx, dev_after_Gy, dev_G, dev_nms, nx, ny);

    hipMemcpy(nms, dev_nms, memSize, hipMemcpyDeviceToHost);

    // edges with nms >= tmax
    memset(h_odata, 0, sizeof(pixel_t) * nx * ny);
    first_edges(nms, h_odata, nx, ny, tmax);

    // edges with nms >= tmin && neighbor is edge
    bool changed;
    do {
        changed = false;
        hysteresis_edges(nms, h_odata, nx, ny, tmin, &changed);
    } while (changed==true);

    hipFree(dev_h_odata);
    hipFree(dev_G);
    hipFree(dev_after_Gx);
    hipFree(dev_after_Gy);
    hipFree(dev_nms);
    hipFree(dev_grad);

    free(nms);
}

// print command line format
void usage(char *command) 
{
    printf("Usage: %s [-h] [-d device] [-i inputfile] [-o outputfile] [-r referenceFile] [-w windowsize] [-t threshold]\n",command);
}

// main
int main( int argc, char** argv) 
{

    // default command line options
    int deviceId = 0;
    char *fileIn=(char *)"lena.pgm",*fileOut=(char *)"lenaOut.pgm",*referenceOut=(char *)"reference.pgm";
    int tmin = 45, tmax = 50;
    float sigma=1.0f; 

    // parse command line arguments
    int opt;
    while( (opt = getopt(argc,argv,"d:i:o:r:n:x:s:h")) !=-1)
    {
        switch(opt)
        {

            case 'd':  // device
                if(sscanf(optarg,"%d",&deviceId)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;

            case 'i': // input image filename
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }

                fileIn = strdup(optarg);
                break;
            case 'o': // output image (from device) filename 
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                fileOut = strdup(optarg);
                break;
            case 'r': // output image (from host) filename
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                referenceOut = strdup(optarg);
                break;
            case 'n': // tmin
                if(strlen(optarg)==0 || sscanf(optarg,"%d",&tmin)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 'x': // tmax
                if(strlen(optarg)==0 || sscanf(optarg,"%d",&tmax)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 's': // sigma
                if(strlen(optarg)==0 || sscanf(optarg,"%f",&sigma)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 'h': // help
                usage(argv[0]);
                exit(0);
                break;

        }
    }

    // select cuda device
    cutilSafeCall( hipSetDevice( deviceId ) );
    
    // create events to measure host canny detector time and device canny detector time
    hipEvent_t startH, stopH, startD, stopD;
    hipEventCreate(&startH);
    hipEventCreate(&stopH);
    hipEventCreate(&startD);
    hipEventCreate(&stopD);

    // allocate host memory
    int* h_idata=NULL;
    unsigned int h,w;

    //load pgm
    if (cutLoadPGMi(fileIn, (unsigned int **)&h_idata, &w, &h) != CUTTrue) {
        printf("Failed to load image file: %s\n", fileIn);
        exit(1);
    }

    // allocate mem for the result on host side
    //int* h_odata = (int*) malloc( h*w*sizeof(unsigned int));
    //int* reference = (int*) malloc( h*w*sizeof(unsigned int));
    
    int* h_odata = (int*) calloc(h*w, sizeof(unsigned int));
    int* reference = (int*) calloc(h*w, sizeof(unsigned int));

    // detect edges at host
    hipEventRecord( startH, 0 );
    cannyHost(h_idata, w, h, tmin, tmax, sigma, reference);   
    hipEventRecord( stopH, 0 ); 
    hipEventSynchronize( stopH );

    // detect edges at GPU
    hipEventRecord( startD, 0 );
    cannyDevice(h_idata, w, h, tmin, tmax, sigma, h_odata);   
    hipEventRecord( stopD, 0 ); 
    hipEventSynchronize( stopD );
    
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    float timeH, timeD;
    hipEventElapsedTime( &timeH, startH, stopH );
    printf( "Host processing time: %f (ms)\n", timeH);
    hipEventElapsedTime( &timeD, startD, stopD );
    printf( "Device processing time: %f (ms)\n", timeD);

    // save output images
    if (cutSavePGMi(referenceOut, (unsigned int *)reference, w, h) != CUTTrue) {
        printf("Failed to save image file: %s\n", referenceOut);
        exit(1);
    }
    if (cutSavePGMi(fileOut,(unsigned int *) h_odata, w, h) != CUTTrue) {
        printf("Failed to save image file: %s\n", fileOut);
        exit(1);
    }

    // cleanup memory
    cutFree( h_idata);
    free( h_odata);
    free( reference);

    cutilDeviceReset();
}
